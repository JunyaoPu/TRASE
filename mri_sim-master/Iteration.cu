#include "hip/hip_runtime.h"
//Main simulator library.
#include "master_def.h"

//Specific coil, sequence... for this simulation.
#include <iostream>
#include "sequence/GRE.cuh"
#include "coil/coil_ideal.cuh"
#include "scanner/scanner.cuh"
#include "primitives/CylinderXY.cuh"
#include "primitives/Box.cuh"
#include "params/simuParams.cuh"
#include "util/recorder.h"
#include "util/vector3.cuh"


#include <time.h>
#include "params/TRASE_Params.cuh"

#include "primitives/Box.cuh"

void wait ( int seconds )
{
  clock_t endwait;
  endwait = clock () + seconds * CLOCKS_PER_SEC ;
  while (clock() < endwait) {}
}


void iteration(real _num){

	//Simulation properties.
	int num_par = 102400;

	SimuParams test_params(num_par, //Number of particles.
		num_par,					//Number of particles per stream.
		8,						//Sequence repeat time.
		0.5,						//Sequence echo time.
		0.001,						//Simulation timestep.
		0,							//Number of particles to track continual, individual magnetization.
		Vector3(0, 0, 1),			//Initial magnetization vector.
		Vector3(0, 0, 0.001),		//Main B0 field direction / strength.
		65,							//(vertical) resolution.
		65,							//(horizontal) resolution.
		5,							//(vertical) FOV.
		5,							//(horizontal) FOV.
		1.005
		);

	TRASE_Params test_TRASE(&test_params);
	Coil_Ideal test_coil;
	GRE test_sequence(&test_params);




//two samples
////////////////////////////////////////////////////////////////////////////////////////////////////////////
/*
	Lattice test_lattice(5.0, 5.0, 5.0, 0.0, 0.0, 0, 2);
	Scanner test_scanner(test_sequence, test_coil, test_params, test_lattice,test_TRASE);

	Cylinder_XY test_primitive(Vector3(-1, 0, 0), 0.9, 0.2, 0.0, 0.0, 0.0/1000.0, 1, 0, num_par);
	test_scanner.add_primitive(test_primitive);
	Cylinder_XY test_primitive_1(Vector3(1, 0, 0), 0.9, 0.2, 0.0, 0.0, 0.02 , 1, 0, num_par);		//the diffusion coefficient must be a float point
	test_scanner.add_primitive(test_primitive_1);
*/


//one sample
						//x,y,z
	//Lattice test_lattice(3.0, 3.0, 0.5, 0.0, 0.0, 0, 1);
	Lattice test_lattice(5.0, 5.0, 5.0, 0.0, 0.0, 0, 1);

	Scanner test_scanner(test_sequence, test_coil, test_params, test_lattice,test_TRASE);
//	Cylinder_XY test_primitive(Vector3(0.0,0.0,0.0), 2, 2, 0.0, 0.0, 0.0/1000.0, 1, 0, num_par);

	Cylinder_XY test_primitive(Vector3(0.0,0.0,0.0), 4.0, 2.0, 20.0*4, 20.0*4, 0.0/1000.0, 1, 0, num_par);			//1.386

	test_scanner.add_primitive(test_primitive);



//////////////////////////////////////////////////////////////////////////////////////////////////////////














	//single sample
	/*
/////////////////////////////////////////////////////////////////////////////////////////////////////////
	Scanner test_scanner(test_sequence, test_coil, test_params,test_TRASE);

	Cylinder_XY test_primitive(Vector3(0.0,0.0,0.0), 4.0, 2.0, 9999.0, 9999.0, 0.0/1000.0, 0, 0, num_par);			//1.386
	test_scanner.add_primitive(test_primitive);
/////////////////////////////////////////////////////////////////////////////////////////////////////////
	*/



	//GPU kernel
	test_scanner.scan();

	//CPU kernel
//	test_scanner.scanCPU();

	hipDeviceSynchronize();
	hipDeviceReset();


}

int main(){

	iteration(0);


	return 0;
}
